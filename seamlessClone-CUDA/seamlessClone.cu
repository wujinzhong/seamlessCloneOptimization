#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2018 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This sample needs at least CUDA 10.0. It demonstrates usages of the nvJPEG
// library nvJPEG supports single and multiple image(batched) decode. Multiple
// images can be decoded using the API for batch mode

#include "opencv2/core.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/videoio.hpp"
#include "opencv2/photo.hpp"
#include<iostream>
#include<string>
#include <hip/hip_runtime_api.h>
#include "nvJPEG_helper.hxx"
#include "seamlessClone.cpp"

using namespace std;
using namespace cv;

struct seamlessClone_params_t {
  int dev;
  hipStream_t stream;
};

int load_inputs( Mat& dst, Mat& patch, Mat& mask,
 string dst_file, string patch_file, string mask_file)
 {
    dst = readFromYaml(dst_file.c_str());
    patch = readFromYaml(patch_file.c_str());
    mask = readFromYaml(mask_file.c_str());

    assert( patch.rows==mask.rows &&
     patch.rows==mask.rows &&
     patch.channels()==3 &&
     mask.channels()==1 );

     return 0;
 }

// parse parameters
int findParamIndex(const char **argv, int argc, const char *parm) {
  int count = 0;
  int index = -1;

  for (int i = 0; i < argc; i++) {
    if (strncmp(argv[i], parm, 100) == 0) {
      index = i;
      count++;
    }
  }

  if (count == 0 || count == 1) {
    return index;
  } else {
    std::cout << "Error, parameter " << parm
              << " has been specified more than once, exiting\n"
              << std::endl;
    return -1;
  }

  return -1;
}

void writeSCImage( const char* writeto, SCImage* img, float scale )
{
	if( img->mDType==SCImageDataType_UC )
	{
		writeBMP( writeto,
		  	img->mData + 0*img->mHeight*img->pitch(),
                  	img->pitch(),
                  	img->mData + 1*img->mHeight*img->pitch(),
                  	img->pitch(),
                  	img->mData + 2*img->mHeight*img->pitch(),
                  	img->pitch(),
                  	img->mWidth,
                  	img->mHeight);
	}
	else if(img->mDType==SCImageDataType_Float)
	{
		SCImage tmp;
		tmp.resize( img->mWidth, img->mHeight, img->mChannel, SCImageDataType_UC, SCImageOrder_Row );
		MyNPP myNpp;
		myNpp.convertFloat2UC( &tmp, img, scale );
		writeSCImage( writeto, &tmp, -1.0f );
		tmp.destroy();
	}
	return;
}

inline int findSampleId(int argc, const char **argv) {
    int sampleId = 6;
    if (checkCmdLineFlag(argc, argv, "sample")) {
        sampleId = getCmdLineArgumentInt(argc, argv, "sample=");
    }
    return sampleId;
}

int main(int argc, const char *argv[]) {
  int pidx;
  if ((pidx = findParamIndex(argv, argc, "-h")) != -1 ||
      (pidx = findParamIndex(argv, argc, "--help")) != -1) {
    std::cout << "Usage: " << argv[0]
              << " [-device=device_id]"
              << std::endl;
    return EXIT_SUCCESS;
  }

  seamlessClone_params_t params;
  params.dev = 0;
  params.dev = findCudaDevice(argc, argv);
  int sampleId = findSampleId(argc, argv);
  hipDeviceProp_t props;
  checkCudaErrors(hipGetDeviceProperties(&props, params.dev));
  printf("Using GPU %d (%s, %d SMs, %d th/SM max, CC %d.%d, ECC %s)\n",
         params.dev, props.name, props.multiProcessorCount,
         props.maxThreadsPerMultiProcessor, props.major, props.minor,
         props.ECCEnabled ? "on" : "off");

  // stream for decoding
  checkCudaErrors(
      hipStreamCreateWithFlags(&params.stream, hipStreamNonBlocking));

  SeamlessClone *seamlessClone = new SeamlessClone( params.stream, 
		  props.multiProcessorCount, 
		  props.maxThreadsPerMultiProcessor );
  hipEvent_t start, stop;
  checkCudaErrors( hipEventCreate(&start) );
  checkCudaErrors( hipEventCreate(&stop) );

#if SCDEBUG
  const int LOOPS = 1;
#else
  const int LOOPS = 50;
#endif
  int centerX = 800, centerY = 100;

  // align API with OpenCV //////////////////////////////////////////////////////////////
  Mat destMat, patchMat, maskMat, _blend;
  vector<string> ymls = {
  "./images/dst.yml",
  "./images/src_154x100.yml",
  "./images/src_mask_154x100.yml",
  
  "./images/dst.yml",
  "./images/src.yml",
  "./images/src_mask_rect255.yml",

  "./images/dst.yml",
  "./images/src_592x592.yml",
  "./images/src_mask_592x592.yml",

  "./images/dst_4800x2694.yml",
  "./images/src_2400x1552.yml",
  "./images/src_mask_2400x1552.yml",
  };
  int file_idx = sampleId;
  if( file_idx!=3 ) centerY = 450;
  if( file_idx==9 ) { centerX = 2400; centerY = 1347; }
  if( load_inputs(destMat, patchMat, maskMat,
                  ymls[file_idx+0],
                  ymls[file_idx+1],
                  ymls[file_idx+2]))
        return EXIT_FAILURE;
  Point p(centerX, centerY);
  int flags = NORMAL_CLONE;

#if !SCDEBUG
  seamlessClone->seamlessCloneGPU( destMat, patchMat, maskMat, p, _blend, flags ); // warm up
#endif

  int   maskWidth = seamlessClone->ucMask.mWidth,
        maskHeight = seamlessClone->ucMask.mHeight;
  checkCudaErrors(hipStreamSynchronize(params.stream));
  checkCudaErrors( hipEventRecord(start, params.stream) );

  for( int l=0; l<LOOPS; l++ )
  {
  	seamlessClone->seamlessCloneGPU( destMat, patchMat, maskMat, p, _blend, flags );
#if SCDEBUG
	std::stringstream ss;
	if( l==0 )
		ss<<"./output/ucRGB_Output"<<".bmp";
	else
		ss<<"./output/ucRGB_Output"<<l<<".bmp";
	writeSCImage(ss.str().c_str(), &seamlessClone->ucRGB_Output, 0 );
    	imwrite( ss.str().c_str(), _blend );
	if( l<LOOPS-1 ) seamlessClone->ucRGB_Output.setConstant(0, params.stream);
#endif
	checkCudaErrors(hipStreamSynchronize(params.stream));
  }

  checkCudaErrors(hipEventRecord(stop, params.stream));
  checkCudaErrors(hipEventSynchronize(stop));

  float msCompute = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msCompute, start, stop));
  printf( "Compute stage performance time= %.3f msec, patch size=%dx%d\n",
            msCompute/LOOPS, maskWidth, maskHeight);
  //writeSCImage("./output/ucRGB_Output.bmp", &seamlessClone->ucRGB_Output, 0 );
  printf( "total device memory used: %d\n", SCImage::getTotalDeviceMemoryOccupy() );
  checkCudaErrors(hipStreamDestroy(params.stream));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  delete seamlessClone;
  return EXIT_SUCCESS;
}
