#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "seamlessclone_cuda.h"

__global__ void cuda_sum_kernel(float *a, float *b, float *c, size_t size)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    c[idx] = a[idx] + b[idx];
}

extern "C" {

void cuda_sum(float *a, float *b, float *c, size_t size)
{
    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, size * sizeof(float));
    hipMalloc((void **)&d_b, size * sizeof(float));
    hipMalloc((void **)&d_c, size * sizeof(float));

    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    cuda_sum_kernel <<< ceil(size / 256.0), 256 >>> (d_a, d_b, d_c, size);

    hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("cuda_sum() called!\n");
}

}
